#include "hip/hip_runtime.h"
#include <iostream>
#include <filesystem>
#include <unordered_map>
#include "LSH.h"
#include "image_preproc.h"

#include "image_preproc.cuh"
#include "LSH.cuh"

int main(int argc, char *argv[])
{
    // Target directory (pictures)
    std::string folder_path = "./Images";

    int num_hashes = 32, dim = SIZE*SIZE, row = SIZE, col = SIZE;

    /******************* Build Hash Functions Start *******************/

    // Too complex to get the correct random number in the Kernel function
    // Abandon the parallel way to create the Hash functions

    int hash_functions = new int[num_hashes];

    for (int i = 0; i < num_hashes; i++)
    {
        hash_functions[i] = new int[dim];

        srand(time(nullptr) + i);
        for (int j = 0; j < dim; j++)
        {
            hash_functions[i][j] = (rand() % 2 == 0 ? 1 : -1);
        }
    }
    
    /******************* Build Hash Functions End *********************/

    std::unordered_map<std::string, std::vector<int>> mapFileHash;

    for (const auto &entry : std::filesystem::directory_iterator(folder_path))
    {
        // Check if the file is regular
        if (std::filesystem::is_regular_file(entry))
        {
            std::string s = entry.path().string();
            const char *filename = s.c_str();

            int *original_image_arr = new int[dim];
            int *feature_image_arr = new int[dim];

            readImageToArr(filename, original_image_arr);

            /******************* Kernel Soble Feature Start *******************/

            int *d_original_image_arr, *d_feature_image_arr; // device copies

            // allocate the GPU memory space
            hipMalloc((void **)&d_original_image_arr, sizeof(int)*dim); 
            hipMalloc((void **)&d_feature_image_arr, sizeof(int)*dim); 

            // copy source data from CPU memory to GPU memory
            hipMemcpy(d_original_image_arr, original_image_arr, dim, hipMemcpyHostToDevice);

            // Define the size of the Grid and the Block
            dim3 dimBlockSoble(32, 32, 1);
            dim3 dimGridSoble(ceil(row / 32.0), ceil(col / 32.0), 1);

            // Execute the kernel fucntion of Soble
            applySobelKernel<<<dimGridSoble, dimBlockSoble>>>(d_original_image_arr, d_feature_image_arr, SIZE, SIZE);

            // copy result data from GPU memory back to CPU memory
            hipMemcpy(feature_image_arr, d_feature_image_arr, dim, hipMemcpyDeviceToHost);

            // Free the allocated GPU memory
            hipFree(d_original_image_arr);
            hipFree(d_feature_image_arr);

            /******************* Kernel Soble Feature End *********************/

            int hash_value_collector = new int[SIZE];// Equal to the number of blocks
            vector<int> hashes_image;

            /******************* Kernel Compute Hash Start *******************/

            int *d_hash_function, *d_feature_image_arr, *d_hash_value_collector; // device copies

            // allocate the GPU memory space
            hipMalloc((void **)&d_hash_function, sizeof(int)*dim); 
            hipMalloc((void **)&d_feature_image_arr, sizeof(int)*dim); 
            hipMalloc((void **)&d_hash_value_collector, sizeof(int)*SIZE);
            // Define the size of the Grid and the Block
            dim3 dimBlockHashComp(1024, 1, 1);
            dim3 dimGridHashComp(ceil(dim / 1024.0), 1, 1);

            // copy source data from CPU memory to GPU memory
            hipMemcpy(d_feature_image_arr, feature_image_arr, dim, hipMemcpyHostToDevice);
            
            for (int i = 0; i < num_hashes; i++) 
            {
                // copy source data from CPU memory to GPU memory
                hipMemcpy(d_hash_function, hash_functions[i], dim, hipMemcpyHostToDevice);

                // Execute the kernel fucntion of Soble
                computeHashKernel<<<dimGridHashComp, dimBlockHashComp>>>(d_hash_function, d_feature_image_arr, d_hash_value_collector, dim);

                // copy result data from GPU memory back to CPU memory
                hipMemcpy(hash_value_collector, d_hash_value_collector, dim, hipMemcpyDeviceToHost);

                int hash_value = 0;
                for (int j = 0; j < SIZE; j++)
                {
                    // Accumulation of each block
                    hash_value += hash_value_collector[j];
                }
                hashes_image.push_back(hash_value);
            }

            // Free the allocated GPU memory
            hipFree(d_original_image_arr);
            hipFree(d_feature_image_arr);

            /******************* Kernel Compute Hash End *********************/

            std::pair<std::string, std::vector<int>> one_pair(s, hash);

            mapFileHash.insert(one_pair);
        }
    }

    for (long unsigned int i = 0; i < files.size(); i++)
    {
        for (long unsigned int j = i + 1; j < files.size(); j++)
        {
            std::vector<int> hash1 = mapFileHash.at(files[i]);
            std::vector<int> hash2 = mapFileHash.at(files[j]);

            double similarity = lsh.calculateSimilarity(hash1, hash2);

            std::cout << "Similarity between " << files[i] << " and " << files[j] << " : " << similarity << std::endl;
        }
    }

    return 0;
}