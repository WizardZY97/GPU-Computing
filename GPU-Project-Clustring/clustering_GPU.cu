#include "hip/hip_runtime.h"
#include <iostream>
#include <filesystem>
#include <unordered_map>
#include <hip/hip_runtime.h>

#include "image_preproc.h"
#include "kernel_func.cuh"

int main(int argc, char *argv[])
{
    // Utilities
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Target directory (pictures)
    std::string folder_path = "./Images";

    int num_hashes = 32, dim = SIZE*SIZE;

    /******************* Build Hash Functions Start *******************/

    // Too complex to get the correct random number in the Kernel function
    // Abandon the parallel way to create the Hash functions

    int **hash_functions = new int*[num_hashes];

    for (int i = 0; i < num_hashes; i++)
    {
        hash_functions[i] = new int[dim];

        srand(time(nullptr) + i);
        for (int j = 0; j < dim; j++)
        {
            hash_functions[i][j] = (rand() % 2 == 0 ? 1 : -1);
        }
    }
    
    /******************* Build Hash Functions End *********************/

    std::unordered_map<std::string, std::vector<int>> mapFileHash;
    std::vector<std::string> files;

    for (const auto &entry : std::filesystem::directory_iterator(folder_path))
    {
        // Check if the file is regular
        if (std::filesystem::is_regular_file(entry))
        {
            std::string s = entry.path().string();
            const char *filename = s.c_str();

            /******************* Kernel Soble Feature Start *******************/

            int *original_image_arr = new int[dim];
            int *feature_image_arr = new int[dim];

            readImageToArr(filename, original_image_arr);

            int *d_original_image_arr, *d_feature_image_arr; // device copies

            // allocate the GPU memory space
            hipMalloc((void **)&d_original_image_arr, sizeof(int)*dim); 
            hipMalloc((void **)&d_feature_image_arr, sizeof(int)*dim); 

            // copy source data from CPU memory to GPU memory
            hipMemcpy(d_original_image_arr, original_image_arr, sizeof(int)*dim, hipMemcpyHostToDevice);

            // Define the size of the Grid and the Block
            dim3 dimBlockSoble(32, 32, 1);
            dim3 dimGridSoble(ceil(SIZE / 32.0), ceil(SIZE / 32.0), 1);

            // Execute the kernel fucntion of Soble
            float msecSoble = 0.0;
            hipEventRecord(start);
            applySobelKernel<<<dimGridSoble, dimBlockSoble>>>(d_original_image_arr, d_feature_image_arr, SIZE, SIZE);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&msecSoble, start, stop);

            // copy result data from GPU memory back to CPU memory
            hipMemcpy(feature_image_arr, d_feature_image_arr, sizeof(int)*dim, hipMemcpyDeviceToHost);

            // Free the allocated GPU memory
            hipFree(d_original_image_arr);
            hipFree(d_feature_image_arr);

            /******************* Kernel Soble Feature End *********************/

            /******************* Kernel Compute Hash Start *******************/

            std::vector<int> hashes_image;

            int *d_hash_function, *d_hash_value_collector; // device copies

            // Define the size of the Grid and the Block
            dim3 dimBlockHashComp(1024, 1, 1);
            dim3 dimGridHashComp(ceil(dim / 1024.0), 1, 1);
            
            float msecHash = 0.0;

            for (int i = 0; i < num_hashes; i++) 
            {
                int *hash_value_collector = new int[SIZE];// Equal to the number of blocks
                
                // allocate the GPU memory space
                hipMalloc((void **)&d_hash_function, sizeof(int)*dim); 
                hipMalloc((void **)&d_feature_image_arr, sizeof(int)*dim); 
                hipMalloc((void **)&d_hash_value_collector, sizeof(int)*SIZE);

                // copy source data from CPU memory to GPU memory
                hipMemcpy(d_hash_function, hash_functions[i], sizeof(int)*dim, hipMemcpyHostToDevice);
                hipMemcpy(d_feature_image_arr, feature_image_arr, sizeof(int)*dim, hipMemcpyHostToDevice);

                // Execute the kernel fucntion of Hash
                float temp = 0.0;
                hipEventRecord(start);
                computeHashKernel<<<dimGridHashComp, dimBlockHashComp>>>(d_hash_function, d_feature_image_arr, d_hash_value_collector, dim);
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&temp, start, stop);
                msecHash += temp;

                // copy result data from GPU memory back to CPU memory
                hipMemcpy(hash_value_collector, d_hash_value_collector, sizeof(int)*SIZE, hipMemcpyDeviceToHost);

                int hash_value = 0;
                for (int j = 0; j < SIZE; j++)
                {
                    // Accumulation of each block
                    hash_value += hash_value_collector[j];
                }
                hashes_image.push_back(hash_value);

                // Free the allocated GPU memory
                hipFree(d_hash_function);
                hipFree(d_hash_value_collector);
                hipFree(d_feature_image_arr);

                delete hash_value_collector;
            }

            /******************* Kernel Compute Hash End *********************/

            std::pair<std::string, std::vector<int>> one_pair(s, hashes_image);

            files.push_back(s);
            mapFileHash.insert(one_pair);

            delete original_image_arr;
            delete feature_image_arr;

            std::cout << "Soble Kernel execution time of " << s << ": " << msecSoble << " milliseconds\n";
            std::cout << "Hash Kernel execution time of " << s << ": " << msecHash << " milliseconds\n";
        }
    }

    for (int i = 0; i < num_hashes; i++)
    {
        delete hash_functions[i];
    }
    delete hash_functions;

    float msecCosSim = 0.0;
    for (long unsigned int i = 0; i < files.size(); i++)
    {
        for (long unsigned int j = i + 1; j < files.size(); j++)
        {
            std::vector<int> hash1 = mapFileHash.at(files[i]);
            std::vector<int> hash2 = mapFileHash.at(files[j]);

            float dot_product = 0, norm_hash1 = 0, norm_hash2 = 0;

            int *d_hash1 = nullptr, *d_hash2 = nullptr; // device copies for the input
            float *d_dot_product = nullptr, *d_norm_hash1 = nullptr, *d_norm_hash2 = nullptr;    // device copies for the output

            // allocate the GPU memory space
            hipMalloc((void **)&d_hash1, sizeof(int)*num_hashes);
            hipMalloc((void **)&d_hash2, sizeof(int)*num_hashes);
            hipMalloc((void **)&d_dot_product, sizeof(float));
            hipMalloc((void **)&d_norm_hash1, sizeof(float));
            hipMalloc((void **)&d_norm_hash2, sizeof(float));

            // copy source data from CPU memory to GPU memory
            hipMemcpy(d_hash1, hash1.data(), sizeof(int)*num_hashes, hipMemcpyHostToDevice);
            hipMemcpy(d_hash2, hash2.data(), sizeof(int)*num_hashes, hipMemcpyHostToDevice);
            hipMemcpy(d_dot_product, &dot_product, sizeof(float), hipMemcpyHostToDevice);// To avoid the random initial value 
            hipMemcpy(d_norm_hash1, &norm_hash1, sizeof(float), hipMemcpyHostToDevice);  // To avoid the random initial value 
            hipMemcpy(d_norm_hash2, &norm_hash2, sizeof(float), hipMemcpyHostToDevice);  // To avoid the random initial value 

            // Define the size of the Grid and the Block (perfectly matching)
            dim3 dimBlockSim(32, 1, 1);
            dim3 dimGridSim(ceil(num_hashes / 32.0), 1, 1);

            float temp = 0.0;
            hipEventRecord(start);
            computeCosSimKernel<<<dimGridSim, dimBlockSim>>>(d_hash1, d_hash2, d_dot_product, d_norm_hash1, d_norm_hash2, num_hashes);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&temp, start, stop);
            msecCosSim += temp;

            // copy result data from GPU memory back to CPU memory
            hipMemcpy(&dot_product, d_dot_product, sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&norm_hash1, d_norm_hash1, sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&norm_hash2, d_norm_hash2, sizeof(float), hipMemcpyDeviceToHost);

            // Free the allocated GPU memory
            hipFree(d_hash1);
            hipFree(d_hash2);
            hipFree(d_dot_product);
            hipFree(d_norm_hash1);
            hipFree(d_norm_hash2);

            // Cosine similarity coefficient
            double sim = dot_product / (sqrt(norm_hash1) * sqrt(norm_hash2));

            std::cout << "Similarity between " << files[i] << " and " << files[j] << " : " << sim << std::endl;
        }
    }
    std::cout << "Total Cosine Similarity Kernel execution time: " << msecCosSim << " milliseconds\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}